#include "hip/hip_runtime.h"
#include "group_graph.h"
#include "kernel_helper.h"
#include "ngap_buffer.h"

using namespace ngap_nb;
template <bool unique>
__global__ void
// __launch_bounds__(256, 6)
advanceAndFilterNonBlockingR1Groups(NonBlockingBuffer nblb,
                                    uint8_t *arr_input_streams,
                                    int arr_input_streams_size,
                                    GroupMatchset gms, GroupNodeAttrs gna,
                                    GroupAAS gaas, GroupCsr gcsr) {

  Matchset symbol_set = gms.groups_ms[blockIdx.x];
  uint8_t *node_attrs = gna.groups_node_attrs[blockIdx.x];
  int *always_active_nodes = gaas.groups_always_active_states[blockIdx.x];
  Csr csr = gcsr.groups_csr[blockIdx.x];
  int input_index = blockIdx.y;

  uint blockId = blockIdx.y * gridDim.x + blockIdx.x;

  const int buffer_capacity_per_block = nblb.buffer_capacity_per_block;
  const int data_buffer_fetch_size = nblb.data_buffer_fetch_size;
  const int add_aas_start = nblb.add_aas_start;
  const int add_aas_interval = nblb.add_aas_interval;
  const int input_bound = (input_index + 1) * arr_input_streams_size;
  int *d_buffer;
  int *d_buffer_idx;
  if (blockId < gridDim.x * gridDim.y / 2) {
    d_buffer = nblb.d_buffer + blockId * buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx + blockId * buffer_capacity_per_block;
  } else {
    d_buffer = nblb.d_buffer2 + (blockId - gridDim.x * gridDim.y / 2) *
                                    buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx2 + (blockId - gridDim.x * gridDim.y / 2) *
                                            buffer_capacity_per_block;
  }
  uint *d_buffer_start = nblb.d_buffer_start + blockId;
  uint *d_buffer_end = nblb.d_buffer_end + blockId;
  uint *d_buffer_end_tmp = nblb.d_buffer_end_tmp + blockId;
  // uint *length = nblb.length + blockIdx.y;
  // uint64_t *results = nblb.d_results;
  uint32_t *d_results_i = nblb.d_results_i;
  uint32_t *d_results_v = nblb.d_results_v;
  unsigned long long int *results_size = nblb.d_results_size;
  // todo(tge): reduce table memory size
  int *d_symbol_table = nblb.d_symbol_table +
                        blockIdx.x * (nblb.num_seg * arr_input_streams_size);
  int *newest_idx = nblb.d_newest_idx + blockId;
  if (csr.alwaysActiveNum == 0) {
    *newest_idx = input_bound;
  }

  // int max_depth = 1;
  auto processRealVertexR0 = [&](int rvertex, int riter, int depth) {
    // advance + filter
    if (riter >= input_bound - 1)
      return;
    uint8_t rsymbol = arr_input_streams[riter + 1];
    int rn_start = csr.GetNeighborListOffset(rvertex);
    int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
    // if (csr.GetNeighborListLength(rvertex) > 3)
    //   printf("%d, ", csr.GetNeighborListLength(rvertex));
    // #pragma unroll 4
    while (rn_start < rn_end) {
      int rneighbor = csr.d_column_indices[rn_start++];
      if (symbol_set.test(rneighbor, rsymbol)) {
        if (false) {
          int mask1 =
              __match_any_sync(__activemask(), getResult(rneighbor, riter));
          int leader = __ffs(mask1) - 1;
          if (threadIdx.x % 32 == leader) {
            addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                              *d_buffer_start, d_buffer_end_tmp,
                              buffer_capacity_per_block);
            if (node_attrs[rneighbor] & 0b10)
              addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                         results_size, nblb.results_capacity, nblb.report_off);
          }
        } else {
          addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                            *d_buffer_start, d_buffer_end_tmp,
                            buffer_capacity_per_block);
          if (node_attrs[rneighbor] & 0b10)
            addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                       results_size, nblb.results_capacity, nblb.report_off);
        }
      }
    }
  };

  auto processRealVertexR1 = [&](int rvertex, int riter, int depth,
                                 bool isUnique) {
    // advance + filter
    if (riter >= input_bound - 1)
      return;
    uint8_t rsymbol = arr_input_streams[riter + 1];
    int rn_start = csr.GetNeighborListOffset(rvertex);
    int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
    // if (csr.GetNeighborListLength(rvertex) > 3)
    //   printf("%d, ", csr.GetNeighborListLength(rvertex));
#pragma unroll 2
    while (rn_start < rn_end) {
      int rneighbor = csr.d_column_indices[rn_start++];
      if (symbol_set.test(rneighbor, rsymbol)) {
        if (unique && isUnique) {
          int mask1 =
              __match_any_sync(__activemask(), getResult(rneighbor, riter));
          int leader = __ffs(mask1) - 1;
          if (threadIdx.x % 32 == leader) {
            if (node_attrs[rneighbor] & 0b10)
              addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                         results_size, nblb.results_capacity, nblb.report_off);
            if (__popc(__activemask()) <= nblb.active_threshold) {
              addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                                *d_buffer_start, d_buffer_end_tmp,
                                buffer_capacity_per_block);
            } else {
              processRealVertexR0(rneighbor, riter + 1, depth + 1);
            }
          }
        } else {
          if (node_attrs[rneighbor] & 0b10)
            addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                       results_size, nblb.results_capacity, nblb.report_off);
          if (__popc(__activemask()) <= nblb.active_threshold) {
            addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                              *d_buffer_start, d_buffer_end_tmp,
                              buffer_capacity_per_block);
          } else {
            processRealVertexR0(rneighbor, riter + 1, depth + 1);
          }
        }
      }
    }
  };

  // uint threadIdInGlobal = blockIdx.x * blockDim.x + threadIdx.x;

  while (*d_buffer_start != *d_buffer_end || *newest_idx < input_bound) {

    if (threadIdx.x == 0)
      if (*d_buffer_start == *d_buffer_end && *newest_idx < input_bound) {
        addToBufferSimple(-1, *newest_idx, d_buffer, d_buffer_idx,
                          *d_buffer_start, d_buffer_end_tmp,
                          buffer_capacity_per_block);
      }

    uint length2 =
        (*d_buffer_end - *d_buffer_start + buffer_capacity_per_block) %
        buffer_capacity_per_block;
    uint length = length2;
    if (length > data_buffer_fetch_size)
      length = data_buffer_fetch_size;
    if (length > 0) {
      uint threadId = threadIdx.x;
      while (threadId < length) {
        uint offset = (*d_buffer_start + threadId) % buffer_capacity_per_block;
        int vertex = d_buffer[offset];
        int iter = d_buffer_idx[offset];
        if (iter >= input_index * arr_input_streams_size &&
            iter < input_bound) {
          // Add fake vertices in batch.
          if ((iter == *newest_idx) &&
              !atomicCAS((int *)(d_symbol_table + *newest_idx), 0, 1)) {
            // int old = atomicAdd(d_buffer_end_tmp, 1);
            // assert((old - *d_buffer_start + buffer_capacity_per_block) %
            //              buffer_capacity_per_block <
            //          buffer_capacity_per_block - 128);
            // d_buffer[old % buffer_capacity_per_block] = -1;
            // d_buffer_idx[old % buffer_capacity_per_block] = iter;

            int iter_rank = *newest_idx - input_index * arr_input_streams_size;
            if (iter_rank >= add_aas_start &&
                ((iter_rank - add_aas_start) % add_aas_interval) == 0) {
              int end_idx = *newest_idx + add_aas_interval;
              if (end_idx > input_bound)
                end_idx = input_bound;
              int start_idx = *newest_idx;
              int number_idx = end_idx - start_idx;
              *newest_idx = end_idx;
              uint old = atomicAdd(d_buffer_end_tmp, number_idx);
              assert((old - *d_buffer_start + buffer_capacity_per_block) %
                         buffer_capacity_per_block <
                     buffer_capacity_per_block - 128);
              for (int i = 0; i < number_idx; i++) {
                d_buffer[(old + i) % buffer_capacity_per_block] = -1;
                d_buffer_idx[(old + i) % buffer_capacity_per_block] =
                    start_idx + i;
              }
            }
          }
          // If vertex < 0, add always active nodes, and fiter them.
          // If vertex > 0, do advance and filter.
          if (vertex < 0) {
            uint8_t symbol = arr_input_streams[iter];
            for (int i = 0; i < csr.alwaysActiveNum; i++) {
              int aan = always_active_nodes[i];
              if (symbol_set.test(aan, symbol)) {
                addToBufferSimple(aan, iter, d_buffer, d_buffer_idx,
                                  *d_buffer_start, d_buffer_end_tmp,
                                  buffer_capacity_per_block);
                if (node_attrs[aan] & 0b10)
                  addResult2(aan, iter, d_results_v, d_results_i, results_size,
                             nblb.results_capacity, nblb.report_off);
              }
            }
          } else {
            bool isUnique = ((nblb.unique_frequency <= 0) ||
                             ((offset % nblb.unique_frequency) == 0));
            processRealVertexR1(vertex, iter, 0, isUnique);
          }
        }
        threadId += blockDim.x;
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
      *d_buffer_end = *d_buffer_end_tmp % buffer_capacity_per_block;
      *d_buffer_start = (*d_buffer_start + length) % buffer_capacity_per_block;
    }
    // __threadfence();
    __syncthreads();
  }
}

template __global__ void advanceAndFilterNonBlockingR1Groups<false>(
    NonBlockingBuffer nblb, uint8_t *arr_input_streams,
    int arr_input_streams_size, GroupMatchset gms, GroupNodeAttrs gna,
    GroupAAS gaas, GroupCsr gcsr);

template __global__ void advanceAndFilterNonBlockingR1Groups<true>(
    NonBlockingBuffer nblb, uint8_t *arr_input_streams,
    int arr_input_streams_size, GroupMatchset gms, GroupNodeAttrs gna,
    GroupAAS gaas, GroupCsr gcsr);

template <bool unique>
__global__ void
// __launch_bounds__(256, 6)
advanceAndFilterNonBlockingR2Groups(NonBlockingBuffer nblb,
                                    uint8_t *arr_input_streams,
                                    int arr_input_streams_size,
                                    GroupMatchset gms, GroupNodeAttrs gna,
                                    GroupAAS gaas, GroupCsr gcsr) {
  Matchset symbol_set = gms.groups_ms[blockIdx.x];
  uint8_t *node_attrs = gna.groups_node_attrs[blockIdx.x];
  int *always_active_nodes = gaas.groups_always_active_states[blockIdx.x];
  Csr csr = gcsr.groups_csr[blockIdx.x];
  int input_index = blockIdx.y;

  uint blockId = blockIdx.y * gridDim.x + blockIdx.x;

  const int buffer_capacity_per_block = nblb.buffer_capacity_per_block;
  const int data_buffer_fetch_size = nblb.data_buffer_fetch_size;
  const int add_aas_start = nblb.add_aas_start;
  const int add_aas_interval = nblb.add_aas_interval;
  const int input_bound = (input_index + 1) * arr_input_streams_size;
  int *d_buffer;
  int *d_buffer_idx;
  if (blockId < gridDim.x * gridDim.y / 2) {
    d_buffer = nblb.d_buffer + blockId * buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx + blockId * buffer_capacity_per_block;
  } else {
    d_buffer = nblb.d_buffer2 + (blockId - gridDim.x * gridDim.y / 2) *
                                    buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx2 + (blockId - gridDim.x * gridDim.y / 2) *
                                            buffer_capacity_per_block;
  }
  uint *d_buffer_start = nblb.d_buffer_start + blockId;
  uint *d_buffer_end = nblb.d_buffer_end + blockId;
  uint *d_buffer_end_tmp = nblb.d_buffer_end_tmp + blockId;
  // uint *length = nblb.length + blockIdx.y;
  // uint64_t *results = nblb.d_results;
  uint32_t *d_results_i = nblb.d_results_i;
  uint32_t *d_results_v = nblb.d_results_v;
  unsigned long long int *results_size = nblb.d_results_size;
  // todo(tge): reduce table memory size
  int *d_symbol_table = nblb.d_symbol_table +
                        blockIdx.x * (nblb.num_seg * arr_input_streams_size);
  int *newest_idx = nblb.d_newest_idx + blockId;

  // int max_depth = 2;
  auto processRealVertexR0 = [&](int rvertex, int riter, int depth) {
    // advance + filter
    if (riter >= input_bound - 1)
      return;
    uint8_t rsymbol = arr_input_streams[riter + 1];
    int rn_start = csr.GetNeighborListOffset(rvertex);
    int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
    // #pragma unroll 2
    while (rn_start < rn_end) {
      int rneighbor = csr.d_column_indices[rn_start++];
      if (symbol_set.test(rneighbor, rsymbol)) {
        if (false) {
          int mask1 =
              __match_any_sync(__activemask(), getResult(rneighbor, riter));
          int leader = __ffs(mask1) - 1;
          if (threadIdx.x % 32 == leader) {
            addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                              *d_buffer_start, d_buffer_end_tmp,
                              buffer_capacity_per_block);
            if (node_attrs[rneighbor] & 0b10)
              addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                         results_size, nblb.results_capacity, nblb.report_off);
          }
        } else {
          addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                            *d_buffer_start, d_buffer_end_tmp,
                            buffer_capacity_per_block);
          if (node_attrs[rneighbor] & 0b10)
            addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                       results_size, nblb.results_capacity, nblb.report_off);
        }
      }
    }
  };

  auto processRealVertexR1 = [&](int rvertex, int riter, int depth) {
    // advance + filter
    if (riter >= input_bound - 1)
      return;
    uint8_t rsymbol = arr_input_streams[riter + 1];
    int rn_start = csr.GetNeighborListOffset(rvertex);
    int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
#pragma unroll 2
    while (rn_start < rn_end) {
      int rneighbor = csr.d_column_indices[rn_start++];
      if (symbol_set.test(rneighbor, rsymbol)) {
        if (__popc(__activemask()) <= nblb.active_threshold) {
          if (false) {
            int mask1 =
                __match_any_sync(__activemask(), getResult(rneighbor, riter));
            int leader = __ffs(mask1) - 1;
            if (threadIdx.x % 32 == leader) {
              addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                                *d_buffer_start, d_buffer_end_tmp,
                                buffer_capacity_per_block);
              if (node_attrs[rneighbor] & 0b10)
                addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                           results_size, nblb.results_capacity,
                           nblb.report_off);
            }
          } else {
            addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                              *d_buffer_start, d_buffer_end_tmp,
                              buffer_capacity_per_block);
            if (node_attrs[rneighbor] & 0b10)
              addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                         results_size, nblb.results_capacity, nblb.report_off);
          }
        } else {
          if (node_attrs[rneighbor] & 0b10)
            addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                       results_size, nblb.results_capacity, nblb.report_off);
          processRealVertexR0(rneighbor, riter + 1, depth + 1);
        }
      }
    }
  };

  auto processRealVertexR2 = [&](int rvertex, int riter, int depth,
                                 bool isUnique) {
    // advance + filter
    if (riter >= input_bound - 1)
      return;
    uint8_t rsymbol = arr_input_streams[riter + 1];
    int rn_start = csr.GetNeighborListOffset(rvertex);
    int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
#pragma unroll 4
    while (rn_start < rn_end) {
      int rneighbor = csr.d_column_indices[rn_start++];
      if (symbol_set.test(rneighbor, rsymbol)) {
        if (unique && isUnique) {
          int mask1 =
              __match_any_sync(__activemask(), getResult(rneighbor, riter));
          int leader = __ffs(mask1) - 1;
          if (threadIdx.x % 32 == leader) {
            if (node_attrs[rneighbor] & 0b10)
              addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                         results_size, nblb.results_capacity, nblb.report_off);
            if (__popc(__activemask()) <= nblb.active_threshold) {
              addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                                *d_buffer_start, d_buffer_end_tmp,
                                buffer_capacity_per_block);
            } else {
              processRealVertexR1(rneighbor, riter + 1, depth + 1);
            }
          }
        } else {
          if (node_attrs[rneighbor] & 0b10)
            addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
                       results_size, nblb.results_capacity, nblb.report_off);
          if (__popc(__activemask()) <= nblb.active_threshold) {
            addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
                              *d_buffer_start, d_buffer_end_tmp,
                              buffer_capacity_per_block);

          } else {
            processRealVertexR1(rneighbor, riter + 1, depth + 1);
          }
        }
      }
    }
  };

  //   auto processRealVertex = [&](auto &&processRealVertex, int rvertex,
  //                                int riter, int depth) {
  //     // advance + filter
  //     if (riter >= input_bound - 1)
  //       return;
  //     uint8_t rsymbol = arr_input_streams[riter + 1];
  //     int rn_start = csr.GetNeighborListOffset(rvertex);
  //     int rn_end = rn_start + csr.GetNeighborListLength(rvertex);
  // #pragma unroll 32
  //     while (rn_start < rn_end) {
  //       int rneighbor = csr.d_column_indices[rn_start++];
  //       if (symbol_set.test(rneighbor, rsymbol)) {
  //         if (depth >= max_depth ||
  //             __popc(__activemask()) <= nblb.active_threshold) {
  //           if (unique) {
  //             int mask1 =
  //                 __match_any_sync(__activemask(), getResult(rneighbor,
  //                 riter));
  //             int leader = __ffs(mask1) - 1;
  //             if (threadIdx.x % 32 == leader) {
  //               addToBufferSimple(rneighbor, riter + 1, d_buffer,
  //               d_buffer_idx,
  //                                 *d_buffer_start, d_buffer_end_tmp,
  //                                 buffer_capacity_per_block);
  //               if (node_attrs[rneighbor] & 0b10)
  //                 addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
  //                            results_size, nblb.results_capacity);
  //             }
  //           } else {
  //             addToBufferSimple(rneighbor, riter + 1, d_buffer, d_buffer_idx,
  //                                 *d_buffer_start, d_buffer_end_tmp,
  //                                 buffer_capacity_per_block);
  //               if (node_attrs[rneighbor] & 0b10)
  //                 addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
  //                            results_size, nblb.results_capacity);
  //           }
  //         } else {
  //           if (node_attrs[rneighbor] & 0b10)
  //               addResult2(rneighbor, riter + 1, d_results_v, d_results_i,
  //                          results_size, nblb.results_capacity);
  //           processRealVertex(processRealVertex, rneighbor, riter + 1, depth
  //           + 1);
  //         }
  //       }
  //     }
  //   };

  // uint threadIdInGlobal = blockIdx.x * blockDim.x + threadIdx.x;

  while (*d_buffer_start != *d_buffer_end || *newest_idx < input_bound) {

    if (threadIdx.x == 0)
      if (*d_buffer_start == *d_buffer_end && *newest_idx < input_bound) {
        addToBufferSimple(-1, *newest_idx, d_buffer, d_buffer_idx,
                          *d_buffer_start, d_buffer_end_tmp,
                          buffer_capacity_per_block);
      }

    uint length2 =
        (*d_buffer_end - *d_buffer_start + buffer_capacity_per_block) %
        buffer_capacity_per_block;
    uint length = length2;
    if (length > data_buffer_fetch_size)
      length = data_buffer_fetch_size;
    if (length > 0) {
      uint threadId = threadIdx.x;

      while (threadId < length) {
        uint offset = (*d_buffer_start + threadId) % buffer_capacity_per_block;
        int vertex = d_buffer[offset];
        int iter = d_buffer_idx[offset];
        if (iter >= input_index * arr_input_streams_size &&
            iter < input_bound) {
          // Add fake vertices in batch.
          if ((iter == *newest_idx) &&
              !atomicCAS((int *)(d_symbol_table + *newest_idx), 0, 1)) {
            // int old = atomicAdd(d_buffer_end_tmp, 1);
            // assert((old - *d_buffer_start + buffer_capacity_per_block) %
            //              buffer_capacity_per_block <
            //          buffer_capacity_per_block - 128);
            // d_buffer[old % buffer_capacity_per_block] = -1;
            // d_buffer_idx[old % buffer_capacity_per_block] = iter;

            int iter_rank = *newest_idx - input_index * arr_input_streams_size;
            if (iter_rank >= add_aas_start &&
                ((iter_rank - add_aas_start) % add_aas_interval) == 0) {
              int end_idx = *newest_idx + add_aas_interval;
              if (end_idx > input_bound)
                end_idx = input_bound;
              int start_idx = *newest_idx;
              int number_idx = end_idx - start_idx;
              *newest_idx = end_idx;
              uint old = atomicAdd(d_buffer_end_tmp, number_idx);
              assert((old - *d_buffer_start + buffer_capacity_per_block) %
                         buffer_capacity_per_block <
                     buffer_capacity_per_block - 128);
              for (int i = 0; i < number_idx; i++) {
                d_buffer[(old + i) % buffer_capacity_per_block] = -1;
                d_buffer_idx[(old + i) % buffer_capacity_per_block] =
                    start_idx + i;
              }
            }
          }
          // If vertex < 0, add always active nodes, and fiter them.
          // If vertex > 0 and we didn't met it before, add a fake
          // vertex(-1) for its iteration.
          if (vertex < 0) {
            // d_symbol_table[iter] = 2;
            uint8_t symbol = arr_input_streams[iter];
            for (int i = 0; i < csr.alwaysActiveNum; i++) {
              int aan = always_active_nodes[i];
              if (symbol_set.test(aan, symbol)) {
                addToBufferSimple(aan, iter, d_buffer, d_buffer_idx,
                                  *d_buffer_start, d_buffer_end_tmp,
                                  buffer_capacity_per_block);
                if (node_attrs[aan] & 0b10)
                  addResult2(aan, iter, d_results_v, d_results_i, results_size,
                             nblb.results_capacity, nblb.report_off);
              }
            }
          } else {
            bool isUnique = ((nblb.unique_frequency <= 0) ||
                             ((offset % nblb.unique_frequency) == 0));
            processRealVertexR2(vertex, iter, 0, isUnique);
            // processRealVertex(processRealVertex, vertex, iter, 0);
          }
        }
        threadId += blockDim.x;
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
      *d_buffer_end = *d_buffer_end_tmp % buffer_capacity_per_block;
      *d_buffer_end_tmp = *d_buffer_end_tmp % buffer_capacity_per_block;
      *d_buffer_start = (*d_buffer_start + length) % buffer_capacity_per_block;
    }
    // __threadfence();
    __syncthreads();
  }
}

template __global__ void advanceAndFilterNonBlockingR2Groups<false>(
    NonBlockingBuffer nblb, uint8_t *arr_input_streams,
    int arr_input_streams_size, GroupMatchset gms, GroupNodeAttrs gna,
    GroupAAS gaas, GroupCsr gcsr);

template __global__ void advanceAndFilterNonBlockingR2Groups<true>(
    NonBlockingBuffer nblb, uint8_t *arr_input_streams,
    int arr_input_streams_size, GroupMatchset gms, GroupNodeAttrs gna,
    GroupAAS gaas, GroupCsr gcsr);