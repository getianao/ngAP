#include "hip/hip_runtime.h"
#include "group_graph.h"
#include "kernel_helper.h"
#include "ngap_buffer.h"

using namespace ngap_nb;

template <bool unique, int precompute_depth, bool record_fs>
__global__ void
// __launch_bounds__(256, 6)
advanceAndFilterNonBlockingPrecGroups(NonBlockingBuffer nblb,
                                      uint8_t *arr_input_streams,
                                      int arr_input_streams_size,
                                      GroupMatchset gms, GroupNodeAttrs gna,
                                      GroupAAS gaas, GroupCsr gcsr) {

  Matchset symbol_set = gms.groups_ms[blockIdx.x];
  uint8_t *node_attrs = gna.groups_node_attrs[blockIdx.x];
  int *always_active_nodes = gaas.groups_always_active_states[blockIdx.x];
  Csr csr = gcsr.groups_csr[blockIdx.x];
  int input_index = blockIdx.y;

  uint blockId = blockIdx.y * gridDim.x + blockIdx.x;

  const int buffer_capacity_per_block = nblb.buffer_capacity_per_block;
  const int data_buffer_fetch_size = nblb.data_buffer_fetch_size;
  const int add_aas_start = nblb.add_aas_start;
  const int add_aas_interval = nblb.add_aas_interval;
  const int input_bound = (input_index + 1) * arr_input_streams_size;
  int *d_buffer;
  int *d_buffer_idx;
  if (blockId < gridDim.x * gridDim.y / 2) {
    d_buffer = nblb.d_buffer + blockId * buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx + blockId * buffer_capacity_per_block;
  } else {
    d_buffer = nblb.d_buffer2 + (blockId - gridDim.x * gridDim.y / 2) *
                                    buffer_capacity_per_block;
    d_buffer_idx = nblb.d_buffer_idx2 + (blockId - gridDim.x * gridDim.y / 2) *
                                            buffer_capacity_per_block;
  }
  uint *d_buffer_start = nblb.d_buffer_start + blockId;
  uint *d_buffer_end = nblb.d_buffer_end + blockId;
  uint *d_buffer_end_tmp = nblb.d_buffer_end_tmp + blockId;
  // uint *length = nblb.length + blockId;
  // uint64_t *results = nblb.d_results;
  uint32_t *d_results_i = nblb.d_results_i;
  uint32_t *d_results_v = nblb.d_results_v;
  unsigned long long int *results_size = nblb.d_results_size;
  // todo(tge): reduce table memory size
  int *d_symbol_table = nblb.d_symbol_table +
                        blockIdx.x * (nblb.num_seg * arr_input_streams_size);
  int *newest_idx = nblb.d_newest_idx + blockId;
  if (csr.alwaysActiveNum == 0) {
    *newest_idx = input_bound;
  }
  // int *fakeiter = nblb.d_fakeiter + blockId * arr_input_streams_size;
  // int *fakeiter_size = nblb.d_fakeiter_size + blockId;
  int *fakeiter2 = nblb.d_fakeiter2 + blockId * nblb.d_fakeiter_capacity;
  int *fakeiter_size2 = nblb.d_fakeiter_size2 + blockId;
  // int *cutoffnum = nblb.cutoffnum + blockId;

  // uint threadIdInGlobal = blockIdx.x * blockDim.x + threadIdx.x;

  while (*d_buffer_start != *d_buffer_end || *newest_idx < input_bound) {

    if (threadIdx.x == 0)
      if (*d_buffer_start == *d_buffer_end && *newest_idx < input_bound) {
        addToBufferSimple(-1, *newest_idx, d_buffer, d_buffer_idx,
                          *d_buffer_start, d_buffer_end_tmp,
                          buffer_capacity_per_block);
      }

    uint length2 =
        (*d_buffer_end - *d_buffer_start + buffer_capacity_per_block) %
        buffer_capacity_per_block;
    uint length = length2;
    if (length > data_buffer_fetch_size)
      length = data_buffer_fetch_size;
    if (length > 0) {
      uint threadId = threadIdx.x;
      while (threadId < length) {
        uint offset = (*d_buffer_start + threadId) % buffer_capacity_per_block;
        int vertex = d_buffer[offset];
        int iter = d_buffer_idx[offset];
        if (iter >= input_index * arr_input_streams_size &&
            iter < input_bound) {
          // Add fake vertices in batch.
          if ((iter == *newest_idx) &&
              !atomicCAS((int *)(d_symbol_table + *newest_idx), 0, 1)) {
            // int old = atomicAdd(d_buffer_end_tmp, 1);
            // assert((old - *d_buffer_start + buffer_capacity_per_block) %
            //              buffer_capacity_per_block <
            //          buffer_capacity_per_block - 128);
            // d_buffer[old % buffer_capacity_per_block] = -1;
            // d_buffer_idx[old % buffer_capacity_per_block] = iter;

            int iter_rank = *newest_idx - input_index * arr_input_streams_size;
            if (iter_rank >= add_aas_start &&
                ((iter_rank - add_aas_start) % add_aas_interval) == 0) {
              int end_idx = *newest_idx + add_aas_interval;
              if (end_idx > input_bound)
                end_idx = input_bound;
              int start_idx = *newest_idx;
              int number_idx = end_idx - start_idx;
              *newest_idx = end_idx;
              uint old = atomicAdd(d_buffer_end_tmp, number_idx);
              assert((old - *d_buffer_start + buffer_capacity_per_block) %
                         buffer_capacity_per_block <
                     buffer_capacity_per_block - 128);
              for (int i = 0; i < number_idx; i++) {
                d_buffer[(old + i) % buffer_capacity_per_block] = -1;
                d_buffer_idx[(old + i) % buffer_capacity_per_block] =
                    start_idx + i;
              }
            }
          }
          // If vertex < 0, add always active nodes, and fiter them.
          // If vertex > 0, do advance and filter.
          if (vertex < 0) {
            if (precompute_depth == 0) {
              uint8_t symbol = arr_input_streams[iter];
              for (int i = 0; i < csr.alwaysActiveNum; i++) {
                int aan = always_active_nodes[i];
                if (symbol_set.test(aan, symbol)) {
                  addToBufferSimple(aan, iter, d_buffer, d_buffer_idx,
                                    *d_buffer_start, d_buffer_end_tmp,
                                    buffer_capacity_per_block);
                  if (node_attrs[aan] & 0b10)
                    addResult2(aan, iter, d_results_v, d_results_i,
                               results_size, nblb.results_capacity,
                               nblb.report_off);
                }
              }
            } else {
              for (int pcd = precompute_depth; pcd > 0; pcd--) { // 3 2 1
                if (iter <= input_bound - pcd) {
                  // Add vertices.
                  uint32_t symbol_idx = 0;
                  for (int loop = 0; loop < pcd; loop++) {
                    symbol_idx =
                        256 * symbol_idx + arr_input_streams[iter + loop];
                  }
                  int symbol_real_idx =
                      csr.d_pts[pcd - 1].getVertexSymbolIndex(symbol_idx);
                  if (symbol_real_idx >= 0) {
                    int poffset =
                        csr.d_pts[pcd - 1].d_vertices_offsets[symbol_real_idx];
                    uint plength =
                        csr.d_pts[pcd - 1]
                            .d_vertices_offsets[symbol_real_idx + 1] -
                        csr.d_pts[pcd - 1].d_vertices_offsets[symbol_real_idx];
                    if (pcd == precompute_depth) {
                      if (plength > csr.d_pts[pcd - 1].cutoff) {
                        int oldf2 = atomicAdd(fakeiter_size2, 1);
                        assert(oldf2 < nblb.d_fakeiter_capacity);
                        fakeiter2[oldf2] = iter;
                      }
                      int min_cut = min(csr.d_pts[pcd - 1].cutoff, plength);
                      for (int j = 0; j < min_cut; j++) {
                        int pov = csr.d_pts[pcd - 1].d_vertices[poffset + j];
                        addToBufferSimple(pov, iter + pcd - 1, d_buffer,
                                          d_buffer_idx, *d_buffer_start,
                                          d_buffer_end_tmp,
                                          buffer_capacity_per_block);
                      }
                    } else {
                      for (int j = 0; j < plength; j++) {
                        addToBufferSimple(
                            csr.d_pts[pcd - 1].d_vertices[poffset + j],
                            iter + pcd - 1, d_buffer, d_buffer_idx,
                            *d_buffer_start, d_buffer_end_tmp,
                            buffer_capacity_per_block);
                      }
                    }
                  }
                  // Add result during precomputing.
                  for (int loop = 0; loop < pcd; loop++) {
                    uint32_t symbol_idx2 = 0;
                    for (int loop2 = 0; loop2 < loop + 1; loop2++) {
                      symbol_idx2 =
                          256 * symbol_idx2 + arr_input_streams[iter + loop2];
                    }
                    int symbol_real_idx2 =
                        csr.d_pts[loop].getResultSymbolIndex(symbol_idx2);
                    if (symbol_real_idx2 >= 0) {
                      int roffset =
                          csr.d_pts[loop].d_results_offsets[symbol_real_idx2];
                      uint rlength =
                          csr.d_pts[loop]
                              .d_results_offsets[symbol_real_idx2 + 1] -
                          roffset;
                      for (int r = 0; r < rlength; r++) {
                        addResult2(csr.d_pts[loop].d_results[roffset + r],
                                   iter + loop, d_results_v, d_results_i,
                                   results_size, nblb.results_capacity,
                                   nblb.report_off);
                      }
                    }
                  }
                  break;
                }
              }
            }
          } else {
            // advance + filter
            if (iter < input_bound - 1) {
              bool isUnique = ((nblb.unique_frequency <= 0) ||
                               ((offset % nblb.unique_frequency) == 0));
              uint8_t next_symbol = arr_input_streams[iter + 1];
              int n_start = csr.GetNeighborListOffset(vertex);
              int n_end = n_start + csr.GetNeighborListLength(vertex);
              while (n_start < n_end) {
                int neighbor = csr.d_column_indices[n_start++];
                if (symbol_set.test(neighbor, next_symbol)) {
                  if (unique && isUnique) {
                    int mask1 = __match_any_sync(__activemask(),
                                                 getResult(neighbor, iter));
                    int leader = __ffs(mask1) - 1;
                    if (threadIdx.x % 32 == leader) {
                      addToBufferSimple(neighbor, iter + 1, d_buffer,
                                        d_buffer_idx, *d_buffer_start,
                                        d_buffer_end_tmp,
                                        buffer_capacity_per_block);
                      if (node_attrs[neighbor] & 0b10)
                        addResult2(neighbor, iter + 1, d_results_v, d_results_i,
                                   results_size, nblb.results_capacity,
                                   nblb.report_off);
                    }
                  } else {
                    addToBufferSimple(neighbor, iter + 1, d_buffer,
                                      d_buffer_idx, *d_buffer_start,
                                      d_buffer_end_tmp,
                                      buffer_capacity_per_block);
                    if (node_attrs[neighbor] & 0b10)
                      addResult2(neighbor, iter + 1, d_results_v, d_results_i,
                                 results_size, nblb.results_capacity,
                                 nblb.report_off);
                  }
                }
              }
            }
          }
        }
        threadId += blockDim.x;
      }
    }
    __syncthreads();

    if (precompute_depth != 0) {
      int ssize2 = *fakeiter_size2;
      if (ssize2 > 0) {
        uint threadId = threadIdx.x;
        int pcd = precompute_depth;
        while (threadId < ssize2) {
          int fiter = fakeiter2[threadId];
          uint32_t symbol_idx = 0;
          for (int loop = 0; loop < pcd; loop++)
            symbol_idx = 256 * symbol_idx + arr_input_streams[fiter + loop];
          int symbol_real_idx =
              csr.d_pts[pcd - 1].getVertexSymbolIndex(symbol_idx);
          if (symbol_real_idx >= 0) {
            int fpoffset =
                csr.d_pts[pcd - 1].d_vertices_offsets[symbol_real_idx] +
                csr.d_pts[pcd - 1].cutoff;
            uint fplength =
                csr.d_pts[pcd - 1].d_vertices_offsets[symbol_real_idx + 1] -
                fpoffset;
            for (int i = 0; i < fplength; i++) {
              int pov = csr.d_pts[pcd - 1].d_vertices[fpoffset + i];
              addToBufferSimple(pov, fiter + pcd - 1, d_buffer, d_buffer_idx,
                                *d_buffer_start, d_buffer_end_tmp,
                                buffer_capacity_per_block);
            }
          }
          threadId += blockDim.x;
        }
      }
      __syncthreads();
    }

    if (threadIdx.x == 0) {
      *d_buffer_end = *d_buffer_end_tmp % buffer_capacity_per_block;
      *d_buffer_end_tmp = *d_buffer_end_tmp % buffer_capacity_per_block;
      *d_buffer_start = (*d_buffer_start + length) % buffer_capacity_per_block;
      if (precompute_depth > 0) {
        *fakeiter_size2 = 0;
      }
      if (record_fs) {
        int old = atomicAdd(nblb.d_froniter_end, 1);
        nblb.d_froniter_length[old] = length;
      }
    }
    // __threadfence();
    __syncthreads();
  }
}

#define __advanceAndFilterNonBlockingPrecGroups(T1, T2, T3)                    \
  template __global__ void advanceAndFilterNonBlockingPrecGroups<T1, T2, T3>(  \
      NonBlockingBuffer nblb, uint8_t * arr_input_streams,                     \
      int arr_input_streams_size, GroupMatchset gms, GroupNodeAttrs gna,       \
      GroupAAS gaas, GroupCsr gcsr);

__advanceAndFilterNonBlockingPrecGroups(false, 0, false);
__advanceAndFilterNonBlockingPrecGroups(true, 0, false);
__advanceAndFilterNonBlockingPrecGroups(false, 0, true);
__advanceAndFilterNonBlockingPrecGroups(true, 0, true);
__advanceAndFilterNonBlockingPrecGroups(false, 1, false);
__advanceAndFilterNonBlockingPrecGroups(true, 1, false);
__advanceAndFilterNonBlockingPrecGroups(false, 1, true);
__advanceAndFilterNonBlockingPrecGroups(true, 1, true);
__advanceAndFilterNonBlockingPrecGroups(false, 2, false);
__advanceAndFilterNonBlockingPrecGroups(true, 2, false);
__advanceAndFilterNonBlockingPrecGroups(false, 2, true);
__advanceAndFilterNonBlockingPrecGroups(true, 2, true);
__advanceAndFilterNonBlockingPrecGroups(false, 3, false);
__advanceAndFilterNonBlockingPrecGroups(true, 3, false);
__advanceAndFilterNonBlockingPrecGroups(false, 3, true);
__advanceAndFilterNonBlockingPrecGroups(true, 3, true);
